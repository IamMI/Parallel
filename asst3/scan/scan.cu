#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void
compare(int N, int* input, int* output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    output[index] = (input[index]==input[index+1]);
}



__global__ void
exclusive_kernel(int N, int* output, bool upsweep, int step) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if((index+1)*step*2-1 >= N){
        return;
    }
    if(upsweep){
        output[(index+1)*2*step-1] += output[(index+1)*2*step-1-step];
    }else{
        int t = output[(index+1)*2*step-1-step];
        output[(index+1)*2*step-1-step] = output[(index+1)*2*step-1];
        output[(index+1)*2*step-1] += t;
    }

}

__global__ void 
fill_repeated_result(int N, int* device_flags, int* device_idx, int* device_output){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index<N && device_flags[index]){
        device_output[device_idx[index]] = index;
    }
}
// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
void exclusive_scan(int* input, int N, int* result)
{
    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.

    int rounded_N = nextPow2(N); // Consider the case that N is not a power of 2


    // upsweep phase
    for (int two_d = 1; two_d <= rounded_N/2; two_d*=2) {
        int two_dplus1 = 2*two_d;
        int blocks_num = (rounded_N + (THREADS_PER_BLOCK * two_dplus1) - 1) / (THREADS_PER_BLOCK * two_dplus1);
        // printf("Blocks num: %d, two_d: %d\n", blocks_num, two_d);
        exclusive_kernel<<<blocks_num, THREADS_PER_BLOCK>>>(rounded_N, result, true, two_d);
    }

    hipMemset(&result[rounded_N-1], 0, sizeof(int));

    // downsweep phase
    for (int two_d = rounded_N/2; two_d >= 1; two_d /= 2) {
        int two_dplus1 = 2*two_d;
        int blocks_num = (rounded_N + (THREADS_PER_BLOCK * two_dplus1) - 1) / (THREADS_PER_BLOCK * two_dplus1);
        exclusive_kernel<<<blocks_num, THREADS_PER_BLOCK>>>(rounded_N, result, false, two_d);
    }
}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    // for(int i=0; i<N; i++){
    //     std::cout << resultarray[i] << " ";
    // }
    // std::cout << std::endl;


    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.
    int* device_flags;
    int* device_idx;

    hipMalloc((void **)&device_flags, sizeof(int) * length);
    hipMalloc((void **)&device_idx, sizeof(int) * length);

    compare<<<(length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(length, device_input, device_flags);
    hipDeviceSynchronize();

    exclusive_scan(device_flags, length, device_idx);
    hipDeviceSynchronize();

    fill_repeated_result<<<(length-1+THREADS_PER_BLOCK)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(length, device_flags, device_idx, device_output);
    hipDeviceSynchronize();

    int num;
    hipMemcpy(&num, device_idx + length - 1, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_flags);
    hipFree(device_idx);

    return num; 
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
